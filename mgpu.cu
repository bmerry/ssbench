/* ssbench: benchmarking of sort and scan libraries
 * Copyright (C) 2014  Bruce Merry
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <vector>
#include <string>
#include <cstddef>
#include "algorithms.h"
#include "register.h"
#include "moderngpu.cuh"
#include "cudautils.h"

class mgpu_algorithm
{
private:
    mgpu::ContextPtr ctx;

public:
    template<typename T>
    struct types
    {
        typedef MGPU_MEM(T) vector;
        typedef vector scan_vector;
        typedef vector sort_vector;
    };

    template<typename T>
    void create(MGPU_MEM(T) &out, std::size_t elements) const
    {
        ctx->Malloc<T>(elements).swap(out);
    }

    template<typename T>
    static void copy(const std::vector<T> &src, MGPU_MEM(T) &dst)
    {
        dst->FromHost(src);
    }

    template<typename T>
    static void copy(const MGPU_MEM(T) &src, MGPU_MEM(T) &dst)
    {
        src->ToDevice(dst->get(), dst->Size());
    }

    template<typename T>
    static void copy(const MGPU_MEM(T) &src, std::vector<T> &dst)
    {
        src->ToHost(dst);
    }

    template<typename T>
    static void pre_scan(const MGPU_MEM(T) &src, MGPU_MEM(T) &dst)
    {
    }

    template<typename T>
    void scan(const MGPU_MEM(T) &src, MGPU_MEM(T) &dst) const
    {
        mgpu::Scan<mgpu::MgpuScanTypeExc>(
            src->get(), src->Size(), T(0), mgpu::plus<T>(), (T *) NULL, (T *) NULL, dst->get(), *ctx);
    }

    template<typename K>
    static void pre_sort(MGPU_MEM(K) &keys)
    {
    }

    template<typename K>
    void sort(MGPU_MEM(K) &keys) const
    {
        mgpu::MergesortKeys(keys->get(), keys->Size(), mgpu::less<K>(), *ctx);
    }

    template<typename K, typename V>
    static void pre_sort_by_key(MGPU_MEM(K) &keys, MGPU_MEM(V) &values)
    {
    }

    template<typename K, typename V>
    void sort_by_key(MGPU_MEM(K) &keys, MGPU_MEM(V) &values) const
    {
        mgpu::MergesortPairs(keys->get(), values->get(), keys->Size(), mgpu::less<K>(), *ctx);
    }

    static void finish()
    {
        CUDA_CHECK( hipDeviceSynchronize() );
    }

    static std::string api() { return "mgpu"; }

    explicit mgpu_algorithm(device_info d)
    {
        if (d.type != DEVICE_TYPE_GPU || d.index >= mgpu::CudaDevice::DeviceCount())
            throw device_not_supported();
        ctx = mgpu::CreateCudaDevice(d.index);
    }
};

static register_algorithms<mgpu_algorithm> register_mgpu;
