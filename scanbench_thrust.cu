#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <vector>
#include <string>
#include "scanbench.h"

template<typename T>
class thrust_scan : public scan_algorithm<T>
{
private:
    thrust::device_vector<T> d_a;
    thrust::device_vector<T> d_scan;

public:
    thrust_scan(const std::vector<T> &h_a)
        : scan_algorithm<T>(h_a), d_a(h_a), d_scan(h_a.size())
    {
    }

    virtual std::string name() const override { return "thrust::exclusive_scan"; }
    virtual std::string api() const override { return "thrust"; }
    virtual void finish() override { hipDeviceSynchronize(); }

    virtual void run() override
    {
        thrust::exclusive_scan(d_a.begin(), d_a.end(), d_scan.begin());
    }

    virtual std::vector<T> get() const override
    {
        std::vector<T> ans(d_scan.size());
        thrust::copy(d_scan.begin(), d_scan.end(), ans.begin());
        return ans;
    }
};

template<typename T>
scan_algorithm<T> *make_thrust_scan(const std::vector<T> &h_a)
{
    return new thrust_scan<T>(h_a);
}

template thrust_scan_factory<int>;

template<typename A>
struct algorithm_factory;

template<typename T>>
struct algorithm_factory<thrust_scan<T> >
{
    scan_algorithm *create(const std::vector<T> &h_a)
    {
        return new thrust_scan<T>(h_a);
    }
};

/********************************************************************/

template<typename T>
class thrust_sort : public sort_algorithm<T>
{
private:
    thrust::device_vector<T> d_a;
    thrust::device_vector<T> d_target;

public:
    thrust_sort(const std::vector<T> &h_a)
        : sort_algorithm(h_a), d_a(h_a), d_target(h_a.size())
    {
    }

    virtual std::string name() const override { return "thrust::sort"; }
    virtual std::string api() const override { return "thrust"; }
    virtual void finish() override { hipDeviceSynchronize(); }

    virtual void run() override
    {
        d_target = d_a;
        thrust::sort(d_target.begin(), d_target.end());
    }

    virtual std::vector<T> get() const override
    {
        std::vector<T> ans(d_target.size());
        thrust::copy(d_target.begin(), d_target.end(), ans.begin());
        return ans;
    }
};

template<typename T>
struct algorithm_factory<thrust_sort<T> >
{
    sort_algorithm<T> *create(const std::vector<T> &h_a)
    {
        return new thrust_sort<T>(h_a);
    }
};
