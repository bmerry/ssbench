/* ssbench: benchmarking of sort and scan libraries
 * Copyright (C) 2014  Bruce Merry
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/sort.h>
#include <vector>
#include <string>
#include "algorithms.h"
#include "register.h"
#include "cudautils.h"

class thrust_algorithm
{
public:
    template<typename T>
    struct types
    {
        typedef thrust::device_vector<T> vector;
        typedef vector scan_vector;
        typedef vector sort_vector;
    };

    template<typename T>
    static void create(thrust::device_vector<T> &out, std::size_t elements)
    {
        out.resize(elements);
    }

    template<typename T>
    static void copy(const std::vector<T> &src, thrust::device_vector<T> &dst)
    {
        thrust::copy(src.begin(), src.end(), dst.begin());
    }

    template<typename T>
    static void copy(const thrust::device_vector<T> &src, thrust::device_vector<T> &dst)
    {
        thrust::copy(src.begin(), src.end(), dst.begin());
    }

    template<typename T>
    static void copy(const thrust::device_vector<T> &src, std::vector<T> &dst)
    {
        thrust::copy(src.begin(), src.end(), dst.begin());
    }

    template<typename T>
    static void pre_scan(const thrust::device_vector<T> &src, thrust::device_vector<T> &dst)
    {
    }

    template<typename T>
    static void scan(const thrust::device_vector<T> &src, thrust::device_vector<T> &dst)
    {
        thrust::exclusive_scan(src.begin(), src.end(), dst.begin());
    }

    template<typename K, typename V>
    static void pre_sort_by_key(thrust::device_vector<K> &keys, thrust::device_vector<V> &values) {}

    template<typename K, typename V>
    static void sort_by_key(thrust::device_vector<K> &keys, thrust::device_vector<V> &values)
    {
        thrust::sort_by_key(keys.begin(), keys.end(), values.begin());
    }

    template<typename T>
    static void pre_sort(thrust::device_vector<T> &keys) {}

    template<typename T>
    static void sort(thrust::device_vector<T> &keys)
    {
        thrust::sort(keys.begin(), keys.end());
    }

    static void finish()
    {
        CUDA_CHECK( hipDeviceSynchronize() );
    }

    static std::string api() { return "thrust"; }

    explicit thrust_algorithm(device_info d)
    {
        cuda_set_device(d);
    }
};

static register_algorithms<thrust_algorithm> register_thrust;
